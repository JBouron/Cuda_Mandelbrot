#include "hip/hip_runtime.h"
/*
	Written by JBouron. See generator.h for more details.
*/
#include <stdio.h>
#include "generator.h"

__device__ int colorize(int i, int max_ite){
	int white = 255 << 24/* | 255 << 16 | 255 << 8 | 255*/;
	//if (i == max_ite) return white;
	//else return 0;
	return ((float)i/max_ite)*white;
}

__global__ void compute_fractal(int* pixels, PRECISION shift_x, PRECISION shift_y, int img_w, int img_h, PRECISION zoom, int max_ite){
	/* Position x and y of the point to be tested in the image. */
	int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

	/* Index of the pixel in the pixels array */
        int px_idx = y * img_w + x;

	/* We test if we are out of bounds */
        if (px_idx > img_w * img_h - 1){
                return;
        }else{
		/* Simple algorithm which test if the given point is in the Mandelbrot Set or not. */	
                PRECISION c_r = shift_x + (x - img_w/2)/zoom;
                PRECISION c_i = shift_y + (y - img_h/2)/zoom;
                PRECISION z_r = 0;
                PRECISION z_i = 0;
                PRECISION i = 0;
                do{
                        PRECISION tmp = z_r;
                        z_r = z_r*z_r - z_i*z_i + c_r;
                        z_i = 2*z_i*tmp + c_i;
                        i ++;
                }while (z_r*z_r + z_i*z_i < 4 && i < max_ite);
		
		/* We compute the color of this point. */
                pixels[px_idx] = colorize(i, max_ite);
        }	
}

int* generate(int* pixels, int img_w, int img_h, int max_ite, PRECISION shift_x, PRECISION shift_y, PRECISION zoom_level){
	/* We first test the validity of the arguments. */
	if (img_w > 0 && img_h > 0 && max_ite > 0 && zoom_level > 0 && pixels != NULL){
		/* Allocating space for the pixels. */
		size_t alloc_size = img_w*img_h*sizeof(int);
	
		/* Allocating memory for the pixels, but this time on the device. */
		int* device_pixels;
		if (hipMalloc((void**) &device_pixels, alloc_size) == hipErrorOutOfMemory){
			fprintf(stderr, "An error has occured while allocating on device memory.\n");
			return NULL;
		}

		/* Defining the number of threads per block and the number of blocks. */
		dim3 threadsPerBlock(16, 16);
	        dim3 numBlocks(img_w / threadsPerBlock.x, img_h / threadsPerBlock.y);	

		/* Calling the kernels */
		compute_fractal<<<numBlocks, threadsPerBlock>>>(device_pixels, shift_x, shift_y, img_w, img_h, zoom_level, max_ite);

		/* Waiting for the end of the computation. */
		hipDeviceSynchronize();

		/* Copying the pixels. */
		hipMemcpy(pixels, device_pixels, alloc_size, hipMemcpyDeviceToHost);
		hipFree(device_pixels);

		/*int i = 0;
        	for (i = 0 ; i < img_w*img_h; i ++){
                	if (i % img_w == 0 && i != 0) printf("\n");
               	 	if (pixels[i] == 0) printf(". ");
                	else printf("# ");
       		}
		printf("\n");*/
		return pixels;
	}
	else return NULL;
}


/** Unit Test **/

#define UNITTEST_IMGW 640
#define UNITTEST_IMGH 640
#define UNITTEST_MAXITE 100
#define UNITTEST_SHIFTX 0.001643721971153
#define UNITTEST_SHIFTY 0.822467633298876
#define UNITTEST_ZOOM 6000000000000.0

//Uint32 white = 255 << 24 | 255 << 16 | 255 << 8 | 255;

int main(void){
	int* pixels = (int*)calloc(UNITTEST_IMGW*UNITTEST_IMGH, sizeof(int));
	if (pixels == NULL){
		printf("Alloc error. Test failed.");
		return -1;
	}

	sf::RenderWindow window(sf::VideoMode(UNITTEST_IMGW, UNITTEST_IMGH), "SFML window");	

	PRECISION z = 1.0;

	sf::Image img;
	sf::Texture tex; 
	sf::Sprite sp;
	
	while (z < UNITTEST_ZOOM){
		if (generate(pixels, UNITTEST_IMGW, UNITTEST_IMGH, UNITTEST_MAXITE, UNITTEST_SHIFTX, UNITTEST_SHIFTY, z) == NULL){
			printf("Generate failed\n.");
			return -1;
		}		
	
		img.create(UNITTEST_IMGW, UNITTEST_IMGH, (sf::Uint8*)pixels);
		tex.loadFromImage(img);
		sp.setTexture(tex);
	
		window.clear();	
		window.draw(sp);
		window.display();
		z *= 2 ;
		printf("Zoom = %f\n", z);
		sf::sleep(sf::milliseconds(500));				
	}
	
	return 0;
}
